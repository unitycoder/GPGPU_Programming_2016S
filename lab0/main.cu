#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cctype>
#include <iostream>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}
__device__ char mytoupper(char input){
	if('a' <= input and input <= 'z'){ return input-('a'-'A');}
	else{ return input;}
}

__global__ void ToCapital(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < fsize and input_gpu[idx] != '\n') {
		input_gpu[idx] = mytoupper(input_gpu[idx]);
	}
}

__global__ void SwitchText(char *input_gpu, int fsize) {
	//int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

   // int row = threadIdx.y;
    int col = threadIdx.x;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

//	Matrix Asub = GetSubMatrix(A, blockRow, m);
	__shared__ char As[2];
	As[col] = input_gpu[idx];

	if (idx < fsize and As[0]!='\n' and As[1]!='\n' and As[0]!=' ' and As[1]!=' ') {
		input_gpu[idx] = As[(col+1)%2];
	}
}


int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	printf("Two transformation function implemented:\n");
	printf("0) convert all characters to capitals\n");
	printf("1) switch all pairs of characters\n");
	printf("type the number to choose the function to demo: ");
	int op = 0;
	scanf("%d", &op);

	if(op){
		SwitchText<<<(fsize/2)+1, 2>>>(input_gpu, fsize);
	}
	else{	
		ToCapital<<<(fsize/32)+1, 32>>>(input_gpu, fsize);
	}
	

	puts(text_smem.get_cpu_ro());
	return 0;
}