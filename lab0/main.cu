#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cctype>
#include <iostream>
#include <cstring>
#include "SyncedMemory.h"

/**
*
* nvcc --version: V7.0.27
* compile: nvcc main.cu -std=c++11 
*
**/

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}
__global__ void ToCapital(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if('a' <= input_gpu[idx] and input_gpu[idx] <= 'z'){ //transform lower cases only
		input_gpu[idx] = input_gpu[idx] - ('a'-'A');
	}
	__syncthreads(); //sync before print

}
__device__ int CheckisText(char input){ // return 1 for english chars
	if('a'<=input and input<='z'){ return 1; }
	if('A'<=input and input<='Z'){ return 1; }
	return 0;
}

__global__ void FindnonText(char *input_gpu, int *nontxtList_gpu, int fsize){
	// find the idx of special characters
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(CheckisText(input_gpu[idx])){ 
		 nontxtList_gpu[idx]=fsize;
	}
	else{ // special chars
		nontxtList_gpu[idx]=idx;
	}
	__syncthreads();
}

__global__ void SwitchText(char *temptext_gpu, int fsize) {
	// swap each pairs

	// int blockRow = blockIdx.y;
	// int blockCol = blockIdx.x;
	// int row = threadIdx.y;
    int col = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// store the pair to swap
	__shared__ char As[2];
	// each thread move 1 data into shared memory
	As[col] = temptext_gpu[idx]; 
	__syncthreads();

	if (CheckisText(As[0]) and CheckisText(As[1])) {
		temptext_gpu[idx] = As[(col+1)%2]; // swap
	}
	__syncthreads();
}

int compare (const void * a, const void * b)
{
  return ( *(int*)a - *(int*)b );
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	
	printf("Two transformation implemented:\n");
	printf("0) convert all text to capitals\n");
	printf("1) swap all pairs of characters in all words\n");
	printf("Please enter the # of the transformation to demo: ");
	int op = 0;
	scanf("%d", &op);

	if(!op){ // op==0, convert all text to capitals
		ToCapital<<<(fsize/32)+1, 32>>>(input_gpu, fsize);
	}
	else{ // op==1, swap all pairs of characters in all words

		// find special characters (those aren't english)
		// store their indexs (nontxtList)
		// (parallel with GPU)
		MemoryBuffer<int> nontxtList(fsize+1);
		auto nontxtList_smem = nontxtList.CreateSync(fsize);
		CHECK;
		int *nontxtList_gpu = nontxtList_smem.get_gpu_rw();
		FindnonText<<<(fsize/32)+1, 32>>>(input_gpu, nontxtList_gpu, fsize);

		// sort the indexs in nontxtList
		int *Lptr = (int*)nontxtList_smem.get_cpu_ro();
		std::qsort(Lptr, fsize, sizeof(int), compare);	
		Lptr = (int*)nontxtList_smem.get_cpu_ro();

		// the chars between each two indexs (temptext)
		// are those we want to swap
		MemoryBuffer<char> temptext(fsize+1);
		auto temptext_smem = temptext.CreateSync(fsize);
		CHECK;
		char *temptext_gpu = temptext_smem.get_gpu_rw();
		
		int len = *Lptr;
		int *nextLptr;
		char *inputptr = text_smem.get_cpu_wo();
		
		// input a word to SwitchText() each time
		// SwitchText swap each pair with shared memory of size 2
		while(*Lptr!=fsize){
				strncpy(temptext_smem.get_cpu_wo(), inputptr, len);
				temptext_smem.get_cpu_wo()[len] = '\0';
				temptext_gpu = temptext_smem.get_gpu_rw();
				SwitchText<<<(len/2)+1, 2>>>(temptext_gpu, len);

				// copy the swaped word 
				strncpy(inputptr, temptext_smem.get_cpu_ro(), len);
				inputptr = inputptr+len+1;

				nextLptr = Lptr+1;
				len = *nextLptr - *Lptr - 1;
				Lptr++;
		}
	}
	puts(text_smem.get_cpu_ro());
	return 0;
}