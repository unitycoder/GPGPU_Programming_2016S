#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#define PRECISION 0.0001

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if ((yt < ht) and (xt < wt) and (mask[curt] > 127.0f) ) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if ((0 <= yb) and (yb < hb) and (0 <= xb) and (xb < wb) ) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void initialAxb(const float* mask, const float* background, const float* target, float* A, float* b, float* x, const int ht, const int wt, const int oy, const int ox, const int wb, const int c, int *f){

	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;

	int yoffset, xoffset, yneibor, xneibor;
	int count = 0;
	int pos = 0;

	if ((0 <= yt) and (yt < ht) and (0 <= xt) and (xt < wt) ) {

		if(mask[curt] > 127){
			x[curt] = target[curt*3+c];

			for(int i=0;i<4;i++){
				if(i==0) {yoffset = -1; xoffset = 0; }
				else if(i==1) {yoffset = 0; xoffset = -1; }
				else if(i==2) {yoffset = 0; xoffset = 1; }
				else {yoffset = 1; xoffset = 0; }

				yneibor = yt+yoffset;
				xneibor = xt+xoffset;
			
				if( ! ((yneibor >= 0) && (yneibor < ht) && (xneibor >= 0) && (xneibor < wt))){ 
					continue; 
				}
				else{
					int neibor = yneibor*wt + xneibor;
					count ++;
					pos = curt*5 + i + 1;
					A[pos] = -1;
					b[curt] += (target[curt*3 + c] - target[(neibor)*3 + c]);
				}
				A[curt*5] = count;
			}
		}
		else{
			x[curt] = background[((yt+oy)*wb + (xt+ox))*3 + c];
//			A[curt*5] = 1;
			for(int i=0; i<5; i++) { A[curt*5 + i] = 0; }
			b[curt] = background[((yt+oy)*wb + (xt+ox))*3 + c]; //0;
		}
	}


}


__global__ void jacobiRow(float* x, float* tmpx, const float* A, const float* b, int ht, int wt){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0;

	int yt = idx/wt, xt = idx%wt;
	int yoffset, xoffset, yneibor, xneibor, neibor;


	if(idx < ht*wt){
//		sum = sum+ A[idx*5] * x[idx];

		for(int i=0;i<4;i++){
			if(i==0) {yoffset = -1; xoffset = 0; }
			else if(i==1) {yoffset = 0; xoffset = -1; }
			else if(i==2) {yoffset = 0; xoffset = 1; }
			else {yoffset = 1; xoffset = 0; }

			yneibor = yt + yoffset;
			xneibor = xt + xoffset;

			if( ! ((yneibor >= 0) && (yneibor < ht) && (xneibor >= 0) && (xneibor < wt))){ 
				continue; 
			}

			neibor = yneibor*wt + xneibor;
			sum = sum+ A[idx*5 +i +1] * x[neibor];
		}
		tmpx[idx] = sum;

	}
}

__device__ float myabs(float x){
	if(x < 0){ return -x; }
	return x;
}

__global__ void copy2x(float* x, float* tmpx, float *A, float *b, const float * mask, int ht, int wt, int * f){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	float tmp = 0;

	if(idx < ht*wt){
		if(mask[idx] > 127){
		//x[idx] = b[idx] - tmpx[idx];
			tmp = (b[idx] - tmpx[idx])/A[idx*5];
			if(myabs(x[idx] - tmp) < PRECISION){f[idx] = 0;}
			x[idx] = tmp;
		}
		else{
			f[idx] = 0;
		}
	}
}

__global__ void paste(float* output, float* x, const float* background, int wt, int ht, const int oy, const int ox, int wb, int hb, int c){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;

	if ((yt < ht) and (xt < wt) ) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if ((0 <= yb) and (yb < hb) and (0 <= xb) and (xb < wb) ) {
			output[curb*3+c] = x[curt];
		}
	}
}

int checkdone(int* fcpu, int ht, int wt){
	for(int i=0; i<wt*ht; i++){
		if(fcpu[i] == 0){return 1;}
	}
	return 0;
}

__global__ void set2one(int* ptr, int sz){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < sz){ptr[idx] = 1;}
}

void addpad(float* padmask, const float* mask, int wt, int ht, float* zero){
	
	hipMemcpy(padmask, zero, sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask+1, zero, wt*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask+wt+1, zero, sizeof(float), hipMemcpyDeviceToDevice);

	for(int i=1, j=0; i<ht+1; i++, j++){
		hipMemcpy(padmask + i*(wt+2), zero, sizeof(float), hipMemcpyDeviceToDevice);
		hipMemcpy(padmask + i*(wt+2) +1, mask + j*wt, wt*sizeof(float), hipMemcpyDeviceToDevice);
		hipMemcpy(padmask + (i+1)*(wt+2) -1, zero, sizeof(float), hipMemcpyDeviceToDevice);
	}
	hipMemcpy(padmask + (ht+1) *(wt+2), zero, sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask + (ht+1)*(wt+2) +1, zero, wt*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask + (ht+1+1)*(wt+2) -1, zero, sizeof(float), hipMemcpyDeviceToDevice);
}

void addpadtarget(float* padmask, const float* mask, int wt, int ht){

	hipMemcpy(padmask, mask, 3*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask+3, mask, 3*wt*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask+3*wt+3, mask+3*wt-3, 3*sizeof(float), hipMemcpyDeviceToDevice);

	for(int i=1, j=0; i<ht+1; i++, j++){
		hipMemcpy(padmask + 3*i*(wt+2), mask + 3*j*wt, 3*sizeof(float), hipMemcpyDeviceToDevice);
		hipMemcpy(padmask + 3*i*(wt+2) + 3, mask + 3*j*wt, 3*wt*sizeof(float), hipMemcpyDeviceToDevice);
		hipMemcpy(padmask + 3*(i+1)*(wt+2) - 3, mask + 3*(j+1)*wt - 3, 3*sizeof(float), hipMemcpyDeviceToDevice);
	}
	hipMemcpy(padmask + 3*(ht+1)*(wt+2), mask + 3*(ht-1)*wt, 3*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask + 3*(ht+1)*(wt+2) +3*1, mask + 3*(ht-1)*wt, 3*wt*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(padmask + 3*(ht+1+1)*(wt+2) -3*1, mask + 3*(ht-1+1)*wt - 3*1, 3*sizeof(float), hipMemcpyDeviceToDevice);
}


void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt0, const int ht0,
	const int oy0, const int ox0
)
{
	int wt=wt0+2, ht=ht0+2;
	int oy=oy0-1, ox=ox0-1;

	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);

	float *A;
	hipMalloc((void **) &A, 5*(wt*ht)*sizeof(float));
	float *b;
	hipMalloc((void **) &b, wt*ht*sizeof(float));
	float *x;
	hipMalloc((void **) &x, wt*ht*sizeof(float));
	float *tmpx;
	hipMalloc((void **) &tmpx, wt*ht*sizeof(float));
    int *f;
	hipMalloc((void **) &f, wt*ht*sizeof(int));	


	float* padmask;
	hipMalloc((void **) &padmask, (wt*ht)*sizeof(float));
	float *zero;
	hipMalloc((void **) &zero, (wt*ht)*sizeof(float));
	hipMemset((void*)zero, 0, wt*ht*sizeof(float));

	addpad(padmask, mask, wt0, ht0, zero);


	float* padtarget;
	hipMalloc((void **) &padtarget, 3*(wt*ht)*sizeof(float));
	addpadtarget(padtarget, target, wt0, ht0);

	for(int c=0; c<3; c++){
		hipMemset((void*)A, 0, 5*(wt*ht)*sizeof(float));
		hipMemset((void*)b, 0, wt*ht*sizeof(float));
		hipMemset((void*)x, 0, wt*ht*sizeof(float));
		hipMemset((void*)tmpx, 0, wt*ht*sizeof(float));
		set2one<<<((ht*wt)/32)+1, 32>>>(f, wt*ht);

		int notyet = wt*ht;

		initialAxb<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(padmask, background, padtarget, A, b, x, ht, wt, oy, ox, wb, c, f);
	
//		int iter =0;
		while(notyet != 0){
			jacobiRow<<<((ht*wt)/32)+1, 32>>>(x, tmpx, A, b, ht, wt);
			copy2x<<<((ht*wt)/32)+1, 32>>>(x, tmpx, A, b, padmask, ht, wt, f);

			thrust::device_vector<int> flag_d(f, f + wt*ht);
			notyet = thrust::reduce(thrust::device, flag_d.begin(), flag_d.end());
//			iter++;
		}
//		printf("%d %d\n", c, iter);
		paste<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(output, x, background, wt, ht, oy, ox, wb, hb, c);
	}
	
	

}