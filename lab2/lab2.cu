#include "hip/hip_runtime.h"
#include "lab2.h"
#include <math.h>
#include <cstdio>
#include <cstdint>
#include <cstdlib>

#include <opencv2/opencv.hpp>

static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;
static const int ANGLE = 1;
static const int octs = 5;

static const double freq = (double)1/(double)32;
static const double Ybound = 225.301;
static const double U = 105.247;
static const double V = 149.173;

#define PI 3.14159265
#define E 2.71828182


 __device__ double dirs[256][2]; 
 __device__ double img[H][W];

 __device__ int perm[256] = { 151,160,137,91,90,15, 
		131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23, 
		190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33, 
		88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166, 
		77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244, 
		102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196, 
		135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123, 
		5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42, 
		223,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167, 43,172,9, 
		129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228, 
		251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107, 
		49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127, 4,150,254, 
		138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180 };



__device__ double power(double x, int y){
	double ans = 1;
	for(int i=0; i<y; i++){ ans = ans * x; }
	return ans;
}

__device__ double dblAbs(double x){
	if(x < 0){ return -x; }
	return x;
}

__device__ double surflet(double x, double y, int perX, int perY, int c, int f){
	
	int gridX = (int)x + c%2, gridY = (int)y + c/2;
	int hashed = perm[ (perm[ (gridX%perX)%256 ] + gridY%perY)%256];
	double grad = (x-gridX) * dirs[(hashed + ANGLE*f) % 256][0] + (y-gridY) * dirs[(hashed + ANGLE*f) % 256][1];

	double distX = dblAbs((double)x-gridX), distY = dblAbs((double)y-gridY);	
	double polyX = 1 - 6*power(distX, 5) + 15*power(distX, 4) - 10*power(distX, 3);
	double polyY = 1 - 6*power(distY, 5) + 15*power(distY, 4) - 10*power(distY, 3);

	return polyX * polyY * grad;
}

__device__ double perlin(double x, double y, int perX, int perY, int f){
	return (surflet(x, y, perX, perY, 0, f) + surflet(x, y, perX, perY, 1, f) + surflet(x, y, perX, perY, 2, f) + surflet(x, y, perX, perY, 3, f));
}


__global__ void fBm(int f, uint8_t *intimgptr, double Yb){

	int perX = (int)((double)W*freq), perY = (int)((double)H*freq);
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int xint = idx%W, yint = idx/W;
	double x = xint*freq, y = yint*freq;

	double ans = 0;
	for(int i=0;i<octs;i++){
		ans += power(0.5, i) * perlin(x*power(2, i), y*power(2, i), perX*power(2, i), perY*power(2, i), f);
	}
	
	img[yint][xint] = ans;
	img[yint][xint] = ((255.0 - (255.0+Yb)/2))*img[yint][xint] + ((255.0+Yb)/2);

	intimgptr[yint*W + xint] = (uint8_t)img[yint][xint];
	__syncthreads();
}

__global__ void initdirs(){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	dirs[idx][0] = cos((idx * 2.0 * PI)/256.0);
	dirs[idx][1] = sin((idx * 2.0 * PI)/256.0);

}

struct Lab2VideoGenerator::Impl {
	int t = 1;
	int f = 1;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
	initdirs<<<1, 256>>>();
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

void Lab2VideoGenerator::Generate(uint8_t *yuv) {
	
	uint8_t *intimgptr;
	hipMalloc((void **) &intimgptr, H*W*sizeof(uint8_t));

	fBm<<<((H*W)/32)+1, 32>>>((impl->f), intimgptr, Ybound);
	hipDeviceSynchronize();

	uint8_t *hostimg = (uint8_t *)malloc(H*W*sizeof(uint8_t));
	hipMemcpy(hostimg, intimgptr, H*W*sizeof(uint8_t), hipMemcpyDeviceToHost);
//	cv::imwrite("uint8Result.png", cv::Mat(H, W, CV_8UC1, hostimg));

	hipMemcpy(yuv, hostimg, H*W, hipMemcpyHostToDevice); // Y

//	hipMemset(yuv, 255/NFRAME, W*H);
	hipMemset(yuv+W*H, (uint8_t)U, W*H/4); // U
	hipMemset(yuv+(W*H)+(W*H/4), (uint8_t)V, W*H/4); // V
	//	if((impl->t % 10 == 0)){ ++(impl->f); }
	++(impl->t);
	++(impl->f);
	
}
